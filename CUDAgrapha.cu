// Define graph of work + dependencies

hipGraphCreate(&graph);
hipGraphAddNode(graph, kernel_a, {}, ...);
hipGraphAddNode(graph, kernel_b, { kernel_a }, ...); // Waits for a
hipGraphAddNode(graph, kernel_c, { kernel_a }, ...); // Waits for a
hipGraphAddNode(graph, kernel_d, { kernel_b, kernel_c }, ...); // Waits for b and c

// Instantiate graph and apply optimizations

hipGraphInstantiate(&instance, graph);

// Launch executable graph 100 times

for(int i=0; i<100; i++)
      hipGraphLaunch(instance, stream);
